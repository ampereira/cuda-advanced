#include "hip/hip_runtime.h"
//
// This sample demonstrates how HyperQ allows supporting devices to avoid false
// dependencies between kernels in different streams.
//
// - Devices without HyperQ will run a maximum of two kernels at a time (one
//   kernel_A and one kernel_B).
// - Devices with HyperQ will run up to 32 kernels simultaneously.

#include <stdio.h>

const char *sSDKsample = "hyperQ";

// This subroutine does no real work but runs for at least the specified number
// of clock ticks.
__device__ void clock_block(clock_t *d_o, clock_t clock_count)
{
    clock_t start_clock = clock();

    clock_t clock_offset = 0;

    while (clock_offset < clock_count)
    {
        clock_offset = clock() - start_clock;
    }

    d_o[0] = clock_offset;
}

// We create two identical kernels calling clock_block(), we create two so that
// we can identify dependencies in the profile timeline ("kernel_B" is always
// dependent on "kernel_A" in the same stream).
__global__ void kernel_A(clock_t *d_o, clock_t clock_count)
{
    clock_block(d_o, clock_count);
}
__global__ void kernel_B(clock_t *d_o, clock_t clock_count)
{
    clock_block(d_o, clock_count);
}

// Single-warp reduction kernel (note: this is not optimized for simplicity)
__global__ void sum(clock_t *d_clocks, int N)
{
    __shared__ clock_t s_clocks[32];

    clock_t my_sum = 0;

    for (int i = threadIdx.x ; i < N ; i += blockDim.x)
    {
        my_sum += d_clocks[i];
    }

    s_clocks[threadIdx.x] = my_sum;
    __syncthreads();

    for (int i = warpSize / 2 ; i > 0 ; i /= 2)
    {
        if (threadIdx.x < i)
        {
            s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
        }

        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        d_clocks[0] = s_clocks[0];
    }
}


int main(int argc, char **argv)
{
    int nstreams = 32;          // One stream for each pair of kernels
    float kernel_time = 10;     // Time each kernel should run in ms
    float elapsed_time;
    int cuda_device = 0;

// *************************************************************************************
// *************************************************************************************
    printf("starting %s...\n", sSDKsample);

    // Get number of streams (if overridden on the command line)
        nstreams = 16;    }
// *************************************************************************************
// *************************************************************************************

    // Allocate host memory for the output (reduced to a single value)
    clock_t *a = 0;
    hipHostMalloc((void **)&a, sizeof(clock_t));

    // Allocate device memory for the output (one value for each kernel)
    clock_t *d_a = 0;

    // Allocate and initialise an array of stream handles


// *************************************************************************************
// *************************************************************************************
    // Create CUDA event handles
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    // Target time per kernel is kernel_time ms, clockRate is in KHz
    // Target number of clocks = target time * clock frequency

    // Start the clock
    hipEventRecord(start_event, 0);
// *************************************************************************************
// *************************************************************************************

    // Queue pairs of {kernel_A, kernel_B} in separate streams

// *************************************************************************************
// *************************************************************************************
    // Stop the clock in stream 0 (i.e. all previous kernels will be complete)
    hipEventRecord(stop_event, 0);
// *************************************************************************************
// *************************************************************************************

    // At this point the CPU has dispatched all work for the GPU and can
    // continue processing other tasks in parallel. In this sample we just want
    // to wait until all work is done so we use a blocking hipMemcpy below.

    // Run the sum kernel and copy the result back to host


// *************************************************************************************
// *************************************************************************************
    // stop_event will have been recorded but including the synchronize here to
    // prevent copy/paste errors!
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time, start_event, stop_event);

    printf("Expected time for serial execution of %d sets of kernels is between approx. %.3fs and %.3fs\n", nstreams, (nstreams + 1) * kernel_time / 1000.0f, 2 * nstreams *kernel_time / 1000.0f);
    printf("Expected time for fully concurrent execution of %d sets of kernels is approx. %.3fs\n", nstreams, 2 * kernel_time / 1000.0f);
    printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

// *************************************************************************************
// *************************************************************************************

    // Release resources (including streams)


// *************************************************************************************
// *************************************************************************************
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
    exit(true);
// *************************************************************************************
// *************************************************************************************
}
